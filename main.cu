#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <hip/hip_runtime.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include "ggx.cuh"

__global__ void test_uniform_hemisphere(float3* ret, int n){
	if(blockIdx.x*blockDim.x + threadIdx.x > 0) return;

	RNG rng(0);

	for(int i=0; i<n; i++){
		ret[i] = sample_uniform_hemisphere(rng.uniform(), rng.uniform());
	}
}


__global__ void test_normalized(float* comp, float* cosine, uint32_t n_v, uint32_t n_m, float alpha){
	uint32_t i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i>=n_v) return;

	float th = i*0.5*M_PI/n_v;
	cosine[i] = cos(th);

	comp[i] = GGX::normalization_constraint(make_float3(sin(th), 0, cos(th)), alpha, n_m);
}

__global__ void to_image(uint32_t* pixels, float* g1_projected, float* cosine, const uint32_t w, const uint32_t h){
	uint32_t i = blockIdx.x*blockDim.x + threadIdx.x;
	uint32_t j = blockIdx.y*blockDim.y + threadIdx.y;
	if(i >= w || j >= h) return;

	pixels[j*w + i] = 0xff3a3a3a;
	if( j == (int)(h*(1-cosine[i])) ) pixels[j*w + i] = 0xffbb7722;
	if( j == (int)(h*(1-g1_projected[i])) ) pixels[j*w + i] = 0xff2299bb;
}


int main(){
	const int cores = 32;
	std::vector<hipStream_t> streams(cores);
	for(auto& stream : streams) hipStreamCreate(&stream);

	const uint32_t w = 800;
	const uint32_t h = 400;

	float* g1_projected_buf;
	hipMallocManaged(&g1_projected_buf, cores * w*sizeof(float));

	float* cosine_buf;
	hipMallocManaged(&cosine_buf, cores * w*sizeof(float));

	uint32_t* pixels_buf;
	hipMallocManaged(&pixels_buf, cores * w*h*sizeof(uint32_t));

	#pragma omp parallel for schedule(dynamic)
	for(int i=0; i<cores; i++){
		const auto& stream = streams[i];
		float* g1_projected = g1_projected_buf + i*w;
		float* cosine = cosine_buf + i*w;
		uint32_t* pixels = pixels_buf + i*w*h;

		float delta = 0.05;
		for(float alpha = delta*i; alpha<1.0; alpha += delta*cores){
			test_normalized <<<512, w/512 + 1, 0, stream>>> (g1_projected, cosine, w, 10000, alpha);

			const dim3 threads(16, 16);
			const dim3 blocks(w/threads.x + 1, h/threads.y + 1);
			to_image <<<blocks, threads, 0, stream>>> (pixels, g1_projected, cosine, w, h);

			hipStreamSynchronize(stream);

			std::string out = "result_gpu/projected_area_gpu_" + std::to_string(alpha) + ".png";
			stbi_write_png(out.c_str(), w, h, 4, pixels, w*sizeof(uint32_t));
		}
	}

	hipFree(g1_projected_buf);
	hipFree(cosine_buf);
	hipFree(pixels_buf);
	for(auto& stream : streams)hipStreamDestroy(stream);
}